#include "hip/hip_runtime.h"
/**
 *  The MIT License (MIT)
 *
 *  Copyright (c) 2015 Kyle Hollins Wray, University of Massachusetts
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy of
 *  this software and associated documentation files (the "Software"), to deal in
 *  the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 *  the Software, and to permit persons to whom the Software is furnished to do so,
 *  subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in all
 *  copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 *  FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 *  COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 *  IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 *  CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */


#include <nova/pomdp/utilities/pomdp_expand_gpu.h>

#include <stdio.h>
#include <cstring>
#include <cstdlib>
#include <time.h>
#include <cmath>

#include <nova/error_codes.h>
#include <nova/constants.h>

namespace nova {

int pomdp_expand_construct_belief_gpu(const POMDP *pomdp, unsigned int i, float *b)
{
    for (unsigned int s = 0; s < pomdp->n; s++) {
        b[s] = 0.0f;
    }
    for (unsigned int j = 0; j < pomdp->rz; j++) {
        int s = pomdp->Z[i * pomdp->rz + j];
        if (s < 0) {
            break;
        }
        b[s] = pomdp->B[i * pomdp->rz + j];
    }

    return NOVA_SUCCESS;
}


int pomdp_expand_belief_update_gpu(const POMDP *pomdp, const float *b, unsigned int a,
    unsigned int o, float *bp)
{
    for (unsigned int sp = 0; sp < pomdp->n; sp++) {
        bp[sp] = 0.0f;
    }

    for (unsigned int s = 0; s < pomdp->n; s++) {
        for (unsigned int i = 0; i < pomdp->ns; i++) {
            int sp = pomdp->S[s * pomdp->m * pomdp->ns + a * pomdp->ns + i];
            if (sp < 0) {
                break;
            }

            bp[sp] += pomdp->T[s * pomdp->m * pomdp->ns + a * pomdp->ns + i] * b[s];
        }
    }

    float normalizingConstant = 0.0f;

    for (unsigned int sp = 0; sp < pomdp->n; sp++) {
        bp[sp] *= pomdp->O[a * pomdp->n * pomdp->z + sp * pomdp->z + o];
        normalizingConstant += bp[sp];
    }

    // If the normalizing constant is exceedingly small, within error tolerances, then this is
    // very likely to be an invalid belief. In practice, this arises when there is a probabilistically
    // impossible observation, given the POMDP.
    if (std::fabs(normalizingConstant) < FLT_ERR_TOL) {
        return NOVA_WARNING_INVALID_BELIEF;
    }

    for (unsigned int sp = 0; sp < pomdp->n; sp++) {
        bp[sp] /= normalizingConstant;
    }

    return NOVA_SUCCESS;
}


int pomdp_expand_probability_observation_gpu(const POMDP *pomdp, const float *b,
    unsigned int a, unsigned int o, float &prObs)
{
    prObs = 0.0f;

    for (unsigned int s = 0; s < pomdp->n; s++) {
        float val = 0.0f;

        for (unsigned int i = 0; i < pomdp->ns; i++) {
            int sp = pomdp->S[s * pomdp->m * pomdp->ns + a * pomdp->ns + i];
            if (sp < 0) {
                break;
            }

            val += pomdp->T[s * pomdp->m * pomdp->ns + a * pomdp->ns + i] *
                    pomdp->O[a * pomdp->n * pomdp->z + sp * pomdp->z + o];
        }

        prObs += val * b[s];
    }

    return NOVA_SUCCESS;
}


int pomdp_expand_update_max_non_zero_values_gpu(const POMDP *pomdp, const float *b,
    unsigned int &maxNonZeroValues)
{
    unsigned int numNonZeroValues = 0;
    for (unsigned int s = 0; s < pomdp->n; s++) {
        if (b[s] > 0.0f) {
            numNonZeroValues++;
        }
    }
    if (numNonZeroValues > maxNonZeroValues) {
        maxNonZeroValues = numNonZeroValues;
    }

    return NOVA_SUCCESS;
}


int pomdp_expand_random_gpu(const POMDP *pomdp, unsigned int numThreads, unsigned int numDesiredBeliefPoints,
    unsigned int &maxNonZeroValues, float *&Bnew)
{
    // Ensure the data is valid.
    if (pomdp == nullptr || pomdp->n == 0 || pomdp->ns == 0 || pomdp->m == 0 ||
            pomdp->z == 0 || pomdp->r == 0 || pomdp->rz == 0 ||
            pomdp->S == nullptr || pomdp->T == nullptr || pomdp->O == nullptr || pomdp->R == nullptr ||
            pomdp->Z == nullptr || pomdp->B == nullptr || pomdp->horizon < 1 ||
            numDesiredBeliefPoints <= 1 || Bnew != nullptr) {
        fprintf(stderr, "Error[pomdp_expand_random_gpu]: %s\n", "Invalid arguments.");
        return NOVA_ERROR_INVALID_DATA;
    }

    srand(time(nullptr));

    maxNonZeroValues = 0;

    // Setup the initial belief point.
    float *b0 = new float[pomdp->n];
    pomdp_expand_construct_belief_gpu(pomdp, 0, b0);

    float *b = new float[pomdp->n];
    unsigned int i = 1;

    // Create the new beliefs matrix. The first one is always the initial seed belief.
    Bnew = new float[numDesiredBeliefPoints * pomdp->n];
    memcpy(&Bnew[0 * pomdp->n], b0, pomdp->n * sizeof(float));

    // For each belief point we want to expand. Each step will generate a new trajectory
    // and add the resulting belief point to B.
    while (i < numDesiredBeliefPoints) {
        // Randomly pick a horizon for this trajectory. We do this because some domains transition
        // beliefs away from areas on the (n-1)-simplex, never to return. This ensures many paths
        // are added.
        unsigned int h = (unsigned int)((float)rand() / (float)RAND_MAX * (float)(pomdp->horizon + 1));

        // Setup the belief used in exploration.
        memcpy(b, b0, pomdp->n * sizeof(float));

        // Follow a random trajectory with length equal to this horizon.
        for (unsigned int t = 0; t < h; t++) {
            // Randomly pick an action.
            unsigned int a = (unsigned int)((float)rand() / (float)RAND_MAX * (float)pomdp->m);

            float currentNumber = 0.0f;
            float targetNumber = (float)rand() / (float)RAND_MAX;

            unsigned int o = 0;
            for (unsigned int op = 0; op < pomdp->z; op++) {
                float prObs = 0.0f;
                pomdp_expand_probability_observation_gpu(pomdp, b, a, op, prObs);
                currentNumber += prObs;

                if (currentNumber >= targetNumber) {
                    o = op;
                    break;
                }
            }

            // Follow the belief update equation to compute b' for all state primes s'.
            float *bp = new float[pomdp->n];
            pomdp_expand_belief_update_gpu(pomdp, b, a, o, bp);
            memcpy(b, bp, pomdp->n * sizeof(float));
            delete [] bp;

            // Determine how many non-zero values exist and update rz.
            pomdp_expand_update_max_non_zero_values_gpu(pomdp, b, maxNonZeroValues);

            // Assign the computed belief for this trajectory.
            memcpy(&Bnew[i * pomdp->n], b, pomdp->n * sizeof(float));

            // Stop if we have met the belief point quota.
            i++;
            if (i >= numDesiredBeliefPoints) {
                break;
            }
        }
    }

    delete [] b;
    delete [] b0;

    return NOVA_SUCCESS;
}

}; // namespace nova

