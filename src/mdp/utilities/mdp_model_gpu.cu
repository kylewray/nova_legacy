/**
 *  The MIT License (MIT)
 *
 *  Copyright (c) 2016 Kyle Hollins Wray, University of Massachusetts
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy of
 *  this software and associated documentation files (the "Software"), to deal in
 *  the Software without restriction, including without limitation the rights to
 *  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 *  the Software, and to permit persons to whom the Software is furnished to do so,
 *  subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in all
 *  copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 *  FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 *  COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 *  IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 *  CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */


#include <nova/mdp/utilities/mdp_model_gpu.h>

#include <stdio.h>

#include <nova/error_codes.h>
#include <nova/constants.h>

namespace nova {

int mdp_initialize_gpu(MDP *mdp)
{
    if (mdp == nullptr) {
        fprintf(stderr, "Error[mdp_initialize_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    int result = 0;

    result += mdp_initialize_successors_gpu(mdp);
    result += mdp_initialize_state_transitions_gpu(mdp);
    result += mdp_initialize_rewards_gpu(mdp);
    if (mdp->ng > 0) {
        result += mdp_initialize_goals_gpu(mdp);
    }

    return result;
}


int mdp_uninitialize_gpu(MDP *mdp)
{
    if (mdp == nullptr) {
        fprintf(stderr, "Error[mdp_uninitialize_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    int result = 0;

    result += mdp_uninitialize_successors_gpu(mdp);
    result += mdp_uninitialize_state_transitions_gpu(mdp);
    result += mdp_uninitialize_rewards_gpu(mdp);
    if (mdp->ng > 0) {
        result += mdp_uninitialize_goals_gpu(mdp);
    }

    return result;
}


int mdp_initialize_successors_gpu(MDP *mdp)
{
    // Ensure the data is valid.
    if (mdp == nullptr || mdp->n == 0 || mdp->m == 0 || mdp->ns == 0 || mdp->S == nullptr) {
        fprintf(stderr, "Error[mdp_initialize_successors_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    // Allocate the memory on the device.
    if (hipMalloc(&mdp->d_S, mdp->n * mdp->m * mdp->ns * sizeof(int)) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_successors_gpu]: %s\n",
                "Failed to allocate device-side memory for the successor states.");
        return NOVA_ERROR_DEVICE_MALLOC;
    }

    // Copy the data from the host to the device.
    if (hipMemcpy(mdp->d_S, mdp->S, mdp->n * mdp->m * mdp->ns * sizeof(int),
                    hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_successors_gpu]: %s\n",
                "Failed to copy memory from host to device for the successor states.");
        return NOVA_ERROR_MEMCPY_TO_DEVICE;
    }

    return NOVA_SUCCESS;
}


int mdp_uninitialize_successors_gpu(MDP *mdp)
{
    if (mdp == nullptr) {
        fprintf(stderr, "Error[mdp_uninitialize_successors_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    if (mdp->d_S != nullptr) {
        if (hipFree(mdp->d_S) != hipSuccess) {
            fprintf(stderr, "Error[mdp_uninitialize_successors_gpu]: %s\n",
                    "Failed to free device-side memory for the successor states.");
            return NOVA_ERROR_DEVICE_FREE;
        }
    }
    mdp->d_S = nullptr;

    return NOVA_SUCCESS;
}


int mdp_initialize_state_transitions_gpu(MDP *mdp)
{
    // Ensure the data is valid.
    if (mdp == nullptr || mdp->n == 0 || mdp->m == 0 || mdp->ns == 0 || mdp->T == nullptr) {
        fprintf(stderr, "Error[mdp_initialize_state_transitions_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    // Allocate the memory on the device.
    if (hipMalloc(&mdp->d_T, mdp->n * mdp->m * mdp->ns * sizeof(float)) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_state_transitions_gpu]: %s\n",
                "Failed to allocate device-side memory for the state transitions.");
        return NOVA_ERROR_DEVICE_MALLOC;
    }

    // Copy the data from the host to the device.
    if (hipMemcpy(mdp->d_T, mdp->T, mdp->n * mdp->m * mdp->ns * sizeof(float),
                    hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error[nova_mdp_pbvi_initialize_state_transitions]: %s\n",
                "Failed to copy memory from host to device for the state transitions.");
        return NOVA_ERROR_MEMCPY_TO_DEVICE;
    }

    return NOVA_SUCCESS;
}


int mdp_uninitialize_state_transitions_gpu(MDP *mdp)
{
    if (mdp == nullptr) {
        fprintf(stderr, "Error[mdp_uninitialize_state_transitions_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    if (mdp->d_T != nullptr) {
        if (hipFree(mdp->d_T) != hipSuccess) {
            fprintf(stderr, "Error[mdp_uninitialize_state_transitions_gpu]: %s\n",
                    "Failed to free device-side memory for the state transitions.");
            return NOVA_ERROR_DEVICE_FREE;
        }
    }
    mdp->d_T = nullptr;

    return NOVA_SUCCESS;
}


int mdp_initialize_rewards_gpu(MDP *mdp)
{
    // Ensure the data is valid.
    if (mdp == nullptr || mdp->n == 0 || mdp->m == 0 || mdp->R == nullptr) {
        fprintf(stderr, "Error[mdp_initialize_rewards_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    // Allocate the memory on the device.
    if (hipMalloc(&mdp->d_R, mdp->n * mdp->m * sizeof(float)) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_rewards_gpu]: %s\n",
                "Failed to allocate device-side memory for the rewards.");
        return NOVA_ERROR_DEVICE_MALLOC;
    }

    // Copy the data from the host to the device.
    if (hipMemcpy(mdp->d_R, mdp->R, mdp->n * mdp->m * sizeof(float),
                    hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_rewards_gpu]: %s\n",
                "Failed to copy memory from host to device for the rewards.");
        return NOVA_ERROR_MEMCPY_TO_DEVICE;
    }

    return NOVA_SUCCESS;
}


int mdp_uninitialize_rewards_gpu(MDP *mdp)
{
    if (mdp == nullptr) {
        fprintf(stderr, "Error[mdp_uninitialize_rewards_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    if (mdp->d_R != nullptr) {
        if (hipFree(mdp->d_R) != hipSuccess) {
            fprintf(stderr, "Error[mdp_uninitialize_rewards_gpu]: %s\n",
                    "Failed to free device-side memory for the rewards.");
            return NOVA_ERROR_DEVICE_FREE;
        }
    }
    mdp->d_R = nullptr;

    return NOVA_SUCCESS;
}


int mdp_initialize_goals_gpu(MDP *mdp)
{
    // Ensure the data is valid.
    if (mdp == nullptr || mdp->ng == 0 || mdp->goals == nullptr) {
        fprintf(stderr, "Error[mdp_initialize_goals_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    // Allocate the memory on the device.
    if (hipMalloc(&mdp->d_goals, mdp->ng * sizeof(unsigned int)) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_goals_gpu]: %s\n",
                "Failed to allocate device-side memory for the goals.");
        return NOVA_ERROR_DEVICE_MALLOC;
    }

    // Copy the data from the host to the device.
    if (hipMemcpy(mdp->d_goals, mdp->goals, mdp->ng * sizeof(unsigned int),
                   hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error[mdp_initialize_goals_gpu]: %s\n",
                "Failed to copy memory from host to device for the goals.");
        return NOVA_ERROR_MEMCPY_TO_DEVICE;
    }

    return NOVA_SUCCESS;
}


int mdp_uninitialize_goals_gpu(MDP *mdp)
{
    if (mdp == nullptr) {
        fprintf(stderr, "Error[mdp_uninitialize_goals_gpu]: %s\n", "Invalid input.");
        return NOVA_ERROR_INVALID_DATA;
    }

    if (mdp->d_goals != nullptr) {
        if (hipFree(mdp->d_goals) != hipSuccess) {
            fprintf(stderr, "Error[mdp_uninitialize_goals_gpu]: %s\n",
                    "Failed to free device-side memory for the goals.");
            return NOVA_ERROR_DEVICE_FREE;
        }
    }
    mdp->d_goals = nullptr;

    return NOVA_SUCCESS;
}

}; // namespace nova

